#include "hip/hip_runtime.h"
/**
 * echo512-80 cuda kernel for X16R algorithm
 *
 * tpruvot 2018 - GPL code
 */

#include <stdio.h>
#include <memory.h>

#include "cuda_helper.h"

extern __device__ __device_builtin__ void __threadfence_block(void);

#include "../x11/cuda_x11_aes.cuh"

__device__ __forceinline__ void AES_2ROUND(const uint32_t* __restrict__ sharedMemory,
	uint32_t &x0, uint32_t &x1, uint32_t &x2, uint32_t &x3,
	uint32_t &k0)
{
	uint32_t y0, y1, y2, y3;

	aes_round(sharedMemory,
		x0, x1, x2, x3,
		k0,
		y0, y1, y2, y3);

	aes_round(sharedMemory,
		y0, y1, y2, y3,
		x0, x1, x2, x3);

	k0++;
}

__global__ void shift_rows(uint32_t *W) {
  uint32_t t[4];
  /// 1, 5, 9, 13
  t[0] = W[threadIdx.x +  4];
  t[1] = W[threadIdx.x +  8];
  t[2] = W[threadIdx.x + 24];
  t[3] = W[threadIdx.x + 60];

  W[threadIdx.x +  4] = W[threadIdx.x + 20];
  W[threadIdx.x +  8] = W[threadIdx.x + 40];
  W[threadIdx.x + 24] = W[threadIdx.x + 56];
  W[threadIdx.x + 60] = W[threadIdx.x + 44];

  W[threadIdx.x + 20] = W[threadIdx.x + 36];
  W[threadIdx.x + 40] = t[1];
  W[threadIdx.x + 56] = t[2];
  W[threadIdx.x + 44] = W[threadIdx.x + 28];

  W[threadIdx.x + 28] = W[threadIdx.x + 12];
  W[threadIdx.x + 12] = t[3];
  W[threadIdx.x + 36] = W[threadIdx.x + 52];
  W[threadIdx.x + 52] = t[0];
}

__global__ void mix_columns(uint32_t *W) {
  uint32_t a[4];
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + (threadIdx.y << 4);//* 16;

  a[0] = W[i + j];
  a[1] = W[i + j + 4];
  a[2] = W[i + j + 8];
  a[3] = W[i + j + 12];

  uint32_t ab = a[0] ^ a[1];
  uint32_t bc = a[1] ^ a[2];
  uint32_t cd = a[2] ^ a[3];

  uint32_t t, t2, t3;
  t  = (ab & 0x80808080);
  t2 = (bc & 0x80808080);
  t3 = (cd & 0x80808080);

  uint32_t abx = (t  >> 7) * 27U ^ ((ab^t)  << 1);
  uint32_t bcx = (t2 >> 7) * 27U ^ ((bc^t2) << 1);
  uint32_t cdx = (t3 >> 7) * 27U ^ ((cd^t3) << 1);

  W[i + j] = bc ^ a[3] ^ abx;
  W[i + j +  4] = a[0] ^ cd ^ bcx;
  W[i + j +  8] = ab ^ a[3] ^ cdx;
  W[i + j + 12] = ab ^ a[2] ^ (abx ^ bcx ^ cdx);
}

__device__
static void echo_round(uint32_t* const sharedMemory, uint32_t *W, uint32_t &k0)
{
	#pragma unroll 16
	for (int idx = 0; idx < 16; idx++) {
		AES_2ROUND(sharedMemory, W[(idx << 2) + 0], W[(idx << 2) + 1], W[(idx << 2) + 2], W[(idx << 2) + 3], k0);
	}

  shift_rows<<<1, 4>>>(W);

	dim3 threadsPerBlock(4, 4);
	mix_columns<<<1, threadsPerBlock>>>(W);
}

__device__ __forceinline__
void cuda_echo_round_80(uint32_t *const __restrict__ sharedMemory, uint32_t *const __restrict__ data, const uint32_t nonce, uint32_t *hash)
{
	uint32_t h[29]; // <= 127 bytes input

	#pragma unroll 8
	for (int i = 0; i < 18; i += 2)
		AS_UINT2(&h[i]) = AS_UINT2(&data[i]);
	h[18] = data[18];
	h[19] = cuda_swab32(nonce);
	h[20] = 0x80;
	h[21] =
  h[22] =
  h[23] =
  h[24] =
  h[25] =
  h[26] = 0;
	h[27] = 0x2000000;
	h[28] = 0x280;

	uint32_t k0 = 640; // bitlen
	__shared__ uint32_t W[64];

	#pragma unroll 8
	for (int i = 0; i < 32; i+=4) {
		W[i] = 512; // L
		W[i+1] = 0; // H
		W[i+2] = 0; // X
		W[i+3] = 0;
	}

	uint32_t Z[16];
	#pragma unroll
	for (int i = 0;  i<16; i++) Z[i] = W[i];
	#pragma unroll
	for (int i = 32; i<61; i++) W[i] = h[i - 32];
	#pragma unroll
	for (int i = 61; i<64; i++) W[i] = 0;

	#pragma unroll
	for (int i = 0; i < 10; i++)
		echo_round(sharedMemory, W, k0);

	#pragma unroll 16
	for (int i = 0; i < 16; i++) {
		Z[i] ^= h[i] ^ W[i] ^ W[i + 32];
	}

	#pragma unroll 8
	for (int i = 0; i < 16; i += 2)
		AS_UINT2(&hash[i]) = AS_UINT2(&Z[i]);
}

__device__ __forceinline__
void echo_gpu_init(uint32_t *const __restrict__ sharedMemory)
{
  sharedMemory[threadIdx.x] = d_AES0[threadIdx.x];
  sharedMemory[threadIdx.x + 128] = d_AES0[threadIdx.x + 128];
  sharedMemory[threadIdx.x + 256] = d_AES1[threadIdx.x];
  sharedMemory[threadIdx.x + 384] = d_AES1[threadIdx.x + 128];
  sharedMemory[threadIdx.x + 512] = d_AES2[threadIdx.x];
  sharedMemory[threadIdx.x + 640] = d_AES2[threadIdx.x + 128];
  sharedMemory[threadIdx.x + 768] = d_AES3[threadIdx.x];
  sharedMemory[threadIdx.x + 896] = d_AES3[threadIdx.x + 128];
}

__host__
void x16_echo512_cuda_init(int thr_id, const uint32_t threads)
{
	aes_cpu_init(thr_id);
}

__constant__ static uint32_t c_PaddedMessage80[20];

__host__
void x16_echo512_setBlock_80(void *endiandata)
{
	hipMemcpyToSymbol(HIP_SYMBOL(c_PaddedMessage80), endiandata, sizeof(c_PaddedMessage80), 0, hipMemcpyHostToDevice);
}

__global__ __launch_bounds__(128, 7) /* will force 72 registers */
void x16_echo512_gpu_hash_80(uint32_t threads, uint32_t startNonce, uint64_t *g_hash)
{
	__shared__ uint32_t sharedMemory[1024];

	echo_gpu_init(sharedMemory);
	__threadfence_block();

	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint64_t hashPosition = thread;
		uint32_t *pHash = (uint32_t*)&g_hash[hashPosition<<3];

		cuda_echo_round_80(sharedMemory, c_PaddedMessage80, startNonce + thread, pHash);
	}
}

__host__
void x16_echo512_cuda_hash_80(int thr_id, const uint32_t threads, const uint32_t startNonce, uint32_t *d_hash)
{
	const uint32_t threadsperblock = 128;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	x16_echo512_gpu_hash_80<<<grid, block>>>(threads, startNonce, (uint64_t*)d_hash);
}
